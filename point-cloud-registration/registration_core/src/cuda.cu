#include "hip/hip_runtime.h"
#include <registration_core/cuda.cuh>
#include <iostream>
#include <vector>

__global__ void add(const int *a, const int *b, int *c) {
    c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

void test(int &c) {
    int *a, *b;  // host copies of a, b
    std::vector<int> va;
    int *d_a, *d_b, *d_c;   // device copies of a, b, c
    int size = N * sizeof(int);
    // Alloc space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);
    // Alloc space for host copies of a, b, c and setup input values
    a = (int *)malloc(size);
    b = (int *)malloc(size);
    a[0] = 1;
    b[0] = 1;
    // Copy inputs to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    // Launch add() kernel on GPU with N threads
    add<<<1,N>>>(d_a, d_b, d_c);
    // Copy result back to host
    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
    // Cleanup
    free(a);
    free(b);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}
